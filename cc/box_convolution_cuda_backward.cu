#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>

#include "box_convolution.h" // for `enum class Parameter`

#define BLOCK_SIZE 256
#define NUM_THREADS 1024

using std::min;
using std::max;

namespace gpu {

template <typename T, size_t N>
using CudaAcsr = const at::PackedTensorAccessor32<T, N, torch::RestrictPtrTraits>;

// TODO switch to square blocks
template <bool normalize, bool exact, typename scalar_t>
__global__ void boxConvUpdateGradInputKernel(
    CudaAcsr<scalar_t,3> gradOutputInt, scalar_t * __restrict__ tmpArray,
    const int32_t * __restrict__ xMinInt , const int32_t * __restrict__ xMaxInt ,
    const int32_t * __restrict__ yMinInt , const int32_t * __restrict__ yMaxInt ,
    const scalar_t * __restrict__ xMinFrac, const scalar_t * __restrict__ xMaxFrac,
    const scalar_t * __restrict__ yMinFrac, const scalar_t * __restrict__ yMaxFrac,
    const scalar_t * __restrict__ area, const int nParams) {
    
    int32_t id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id;

    const int32_t h = gradOutputInt.size(1) - 1;
    const int32_t w = gradOutputInt.size(2) - 1;
    const int32_t y = id % w; id /= w;
    const int32_t x = id % h; id /= h;
    const int32_t paramIdx = id % nParams;

    // `id` is now the current plane number
    auto gradOutputIntPlane = gradOutputInt[id];

    if (id < gradOutputInt.size(0)) {

        const int32_t xMinCurr = xMinInt[paramIdx];
        const int32_t xMaxCurr = xMaxInt[paramIdx];
        const int32_t yMinCurr = yMinInt[paramIdx];
        const int32_t yMaxCurr = yMaxInt[paramIdx];

        const int t = max(0, min(x+xMinCurr, h));
        const int b = max(0, min(x+xMaxCurr, h));
        const int l = max(0, min(y+yMinCurr, w));
        const int r = max(0, min(y+yMaxCurr, w));

        scalar_t outValue;

        outValue = 
              gradOutputIntPlane[b][r]
            - gradOutputIntPlane[t][r]
            - gradOutputIntPlane[b][l]
            + gradOutputIntPlane[t][l];

        if (exact) {
            const scalar_t xMinCurrFrac = xMinFrac[paramIdx];
            const scalar_t xMaxCurrFrac = xMaxFrac[paramIdx];
            const scalar_t yMinCurrFrac = yMinFrac[paramIdx];
            const scalar_t yMaxCurrFrac = yMaxFrac[paramIdx];

            const int tAdv = x+xMinCurr-1 <  h ? max(0, min(t-1, h)) : t;
            const int bAdv = x+xMaxCurr   >= 0 ? max(0, min(b+1, h)) : b;
            const int lAdv = y+yMinCurr-1 <  w ? max(0, min(l-1, w)) : l;
            const int rAdv = y+yMaxCurr   >= 0 ? max(0, min(r+1, w)) : r;

            // -- xMax border
            outValue +=
                ( gradOutputIntPlane[bAdv][r]
                - gradOutputIntPlane[b   ][r]
                - gradOutputIntPlane[bAdv][l]
                + gradOutputIntPlane[b   ][l]
                ) * xMaxCurrFrac;

            // -- yMax border
            outValue +=
                ( gradOutputIntPlane[b][rAdv]
                - gradOutputIntPlane[b][r   ]
                - gradOutputIntPlane[t][rAdv]
                + gradOutputIntPlane[t][r   ]
                ) * yMaxCurrFrac;

            // -- xMin border
            outValue +=
                ( gradOutputIntPlane[t   ][r]
                - gradOutputIntPlane[tAdv][r]
                - gradOutputIntPlane[t   ][l]
                + gradOutputIntPlane[tAdv][l]
                ) * xMinCurrFrac;

            // -- yMin border
            outValue +=
                ( gradOutputIntPlane[b][l   ]
                - gradOutputIntPlane[b][lAdv]
                - gradOutputIntPlane[t][l   ]
                + gradOutputIntPlane[t][lAdv]
                ) * yMinCurrFrac;

            // -- corner pixels
            outValue += 
                xMaxCurrFrac*yMaxCurrFrac * (
                   (x+xMaxCurr >= h or
                    y+yMaxCurr >= w or
                    x+xMaxCurr <  0 or
                    y+yMaxCurr <  0 or
                    b == bAdv or
                    r == rAdv) ? static_cast<scalar_t>(0) : 
                    
                    ( gradOutputIntPlane[b+1][r+1]
                    - gradOutputIntPlane[b  ][r+1]
                    - gradOutputIntPlane[b+1][r  ]
                    + gradOutputIntPlane[b  ][r  ]));

            outValue +=
                xMinCurrFrac*yMaxCurrFrac * (
                   (x+xMinCurr >  h or
                    y+yMaxCurr >= w or
                    x+xMinCurr <= 0 or
                    y+yMaxCurr <  0 or
                    t == tAdv or
                    r == rAdv) ? static_cast<scalar_t>(0) : 
                    
                    ( gradOutputIntPlane[tAdv+1][r+1]
                    - gradOutputIntPlane[tAdv+1][r  ]
                    - gradOutputIntPlane[tAdv  ][r+1]
                    + gradOutputIntPlane[tAdv  ][r  ]));

            outValue +=
                xMaxCurrFrac*yMinCurrFrac * (
                   (x+xMaxCurr >= h or
                    y+yMinCurr >  w or
                    x+xMaxCurr <  0 or
                    y+yMinCurr <= 0 or
                    b == bAdv or
                    l == lAdv) ? static_cast<scalar_t>(0) : 
                    
                    ( gradOutputIntPlane[b+1][lAdv+1]
                    - gradOutputIntPlane[b  ][lAdv+1]
                    - gradOutputIntPlane[b+1][lAdv  ]
                    + gradOutputIntPlane[b  ][lAdv  ]));

            outValue +=
                xMinCurrFrac*yMinCurrFrac * (
                   (x+xMinCurr >  h or
                    y+yMinCurr >  w or
                    x+xMinCurr <= 0 or
                    y+yMinCurr <= 0 or
                    t == tAdv or
                    l == lAdv) ? static_cast<scalar_t>(0) : 
                    
                    ( gradOutputIntPlane[tAdv+1][lAdv+1]
                    - gradOutputIntPlane[tAdv+1][lAdv  ]
                    - gradOutputIntPlane[tAdv  ][lAdv+1]
                    + gradOutputIntPlane[tAdv  ][lAdv  ]));
        }

        *tmpArray = outValue * (normalize ? area[paramIdx] : static_cast<scalar_t>(1));
    }
}

template <bool normalize, bool exact>
void boxConvUpdateGradInput(
    at::Tensor & xMinInt , at::Tensor & xMaxInt , at::Tensor & yMinInt , at::Tensor & yMaxInt ,
    at::Tensor & xMinFrac, at::Tensor & xMaxFrac, at::Tensor & yMinFrac, at::Tensor & yMaxFrac,
    at::Tensor & area, at::Tensor & grad_output_integrated, at::Tensor & tmpArray) {

    // TODO use square blocks as in `boxConvUpdateOutput`?
    const int threadsNeeded = tmpArray.numel();
    int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(tmpArray.scalar_type(), "gpu::boxConvUpdateGradInput", ([&] {
        auto gradOutputIntFlattened = grad_output_integrated.view(
            {-1, grad_output_integrated.size(-2), grad_output_integrated.size(-1)});
        auto gradOutputIntAcsr =
            gradOutputIntFlattened.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        boxConvUpdateGradInputKernel <normalize, exact>
            <<<numBlocks, NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>> (
            gradOutputIntAcsr, tmpArray.data_ptr<scalar_t>(),
            xMinInt.data_ptr<int32_t>(), xMaxInt.data_ptr<int32_t>(),
            yMinInt.data_ptr<int32_t>(), yMaxInt.data_ptr<int32_t>(),
            xMinFrac.data_ptr<scalar_t>(), xMaxFrac.data_ptr<scalar_t>(),
            yMinFrac.data_ptr<scalar_t>(), yMaxFrac.data_ptr<scalar_t>(),
            normalize ? area.data_ptr<scalar_t>() : nullptr, xMinInt.numel());
        THCudaCheck(hipGetLastError());
    }));
}

// explicitly instantiate
template void boxConvUpdateGradInput<true, true>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateGradInput<false, true>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateGradInput<true, false>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateGradInput<false, false>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);


// TODO overload for exact/truncated mode
// TODO accept only three pairs of parameter arrays, not four (one is always redundant)
template <Parameter parameter, bool exact, typename scalar_t>
__global__ void boxConvAccGradParametersKernel(
    CudaAcsr<scalar_t,3> inputInt, scalar_t * __restrict__ tmpArray,
    const int32_t * __restrict__ xMinInt , const int32_t * __restrict__ xMaxInt ,
    const int32_t * __restrict__ yMinInt , const int32_t * __restrict__ yMaxInt ,
    const scalar_t * __restrict__ xMinFrac, const scalar_t * __restrict__ xMaxFrac,
    const scalar_t * __restrict__ yMinFrac, const scalar_t * __restrict__ yMaxFrac,
    const int nParams) {
    
    int32_t id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id;

    const int32_t h = inputInt.size(1) - 1;
    const int32_t w = inputInt.size(2) - 1;
    const int32_t y = id % w + 1; id /= w;
    const int32_t x = id % h + 1; id /= h;
    const int32_t paramIdx = id % nParams; id /= nParams;

    // `id` is now the current absolute input plane number
    auto inputIntPlane = inputInt[id];

    if (id < inputInt.size(0)) {

        const int32_t xMinCurr = xMinInt[paramIdx];
        const int32_t xMaxCurr = xMaxInt[paramIdx];
        const int32_t yMinCurr = yMinInt[paramIdx];
        const int32_t yMaxCurr = yMaxInt[paramIdx];

        // TODO only define these if `exact == true`
        const scalar_t xMinCurrFrac = xMinFrac[paramIdx];
        const scalar_t xMaxCurrFrac = xMaxFrac[paramIdx];
        const scalar_t yMinCurrFrac = yMinFrac[paramIdx];
        const scalar_t yMaxCurrFrac = yMaxFrac[paramIdx];

        int valid;
        int cornerX, cornerY;
        
        scalar_t delta = 0;

        if (parameter == Parameter::xMin) {
            if (exact) {
            // TODO maybe use `input` instead of `inputInt`
            valid =
                not (y+yMinCurr < 1) & not (y+yMinCurr > w) & not (x+xMinCurr < 1);
            cornerX = max(0,min(h-1,x+xMinCurr-1));
            cornerY = max(0,min(w-1,y+yMinCurr-1));
            const scalar_t tlCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            valid = 
                not (y+yMaxCurr  < 0) & not (y+yMaxCurr  >= w) & not (x+xMinCurr  < 1);
            cornerX = max(0,min(h-1,x+xMinCurr -1));
            cornerY = max(0,min(w-1,y+yMaxCurr   ));
            const scalar_t trCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            delta += trCorner * yMaxCurrFrac;
            delta += tlCorner * yMinCurrFrac;
            } // if (exact)

            delta += inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMaxCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMinCurr -1, h))][max(0,min(y+yMaxCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMinCurr   , w))];
            delta += inputIntPlane
                [max(0,min(x+xMinCurr -1, h))][max(0,min(y+yMinCurr   , w))];

            delta *= (x+xMinCurr  >= 1) & (x+xMinCurr  <= h);

            *tmpArray = -delta;
        }

        else if (parameter == Parameter::xMax) {
            if (exact) {
            valid =
                not (y+yMinCurr  < 1) & not (y+yMinCurr  > w) & not (x+xMaxCurr  >= h);
            cornerX = max(0,min(h-1,x+xMaxCurr   ));
            cornerY = max(0,min(w-1,y+yMinCurr -1));
            const scalar_t blCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            valid = 
                not (y+yMaxCurr  < 0) & not (y+yMaxCurr  >= w) & not (x+xMaxCurr  >= h);
            cornerX = max(0,min(h-1,x+xMaxCurr   ));
            cornerY = max(0,min(w-1,y+yMaxCurr   ));
            const scalar_t brCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            delta += brCorner * yMaxCurrFrac;
            delta += blCorner * yMinCurrFrac;
            } // if (exact)

            delta += inputIntPlane
                [max(0,min(x+xMaxCurr +1, h))][max(0,min(y+yMaxCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMaxCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMaxCurr +1, h))][max(0,min(y+yMinCurr   , w))];
            delta += inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMinCurr   , w))];

            delta *= (x+xMaxCurr  >= 0) & (x+xMaxCurr  < h);

            *tmpArray = delta;
        }

        else if (parameter == Parameter::yMin) {
            if (exact) {
            valid =
                not (y+yMinCurr  < 1) & not (x+xMinCurr  < 1) & not (x+xMinCurr  > h);
            cornerX = max(0,min(h-1,x+xMinCurr -1));
            cornerY = max(0,min(w-1,y+yMinCurr -1));
            const scalar_t tlCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            valid = 
                not (y+yMinCurr  < 1) & not (x+xMaxCurr  < 0) & not (x+xMaxCurr  >= h);
            cornerX = max(0,min(h-1,x+xMaxCurr   ));
            cornerY = max(0,min(w-1,y+yMinCurr -1));
            const scalar_t blCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            delta += tlCorner * xMinCurrFrac;
            delta += blCorner * xMaxCurrFrac;
            } // if (exact)

            delta += inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMinCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMinCurr -1, w))];
            delta -= inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMinCurr   , w))];
            delta += inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMinCurr -1, w))];

            delta *= (y+yMinCurr  >= 1) & (y+yMinCurr  <= w);

            *tmpArray = -delta;
        }

        else if (parameter == Parameter::yMax) {
            if (exact) {
            valid =
                not (y+yMaxCurr  >= w) & not (x+xMinCurr  < 1) & not (x+xMinCurr  > h);
            cornerX = max(0,min(h-1,x+xMinCurr -1));
            cornerY = max(0,min(w-1,y+yMaxCurr   ));
            const scalar_t trCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            valid = 
                not (y+yMaxCurr  >= w) & not (x+xMaxCurr  < 0) & not (x+xMaxCurr  >= h);
            cornerX = max(0,min(h-1,x+xMaxCurr   ));
            cornerY = max(0,min(w-1,y+yMaxCurr   ));
            const scalar_t brCorner = valid * 
                ( inputIntPlane[cornerX+1][cornerY+1]
                - inputIntPlane[cornerX  ][cornerY+1]
                - inputIntPlane[cornerX+1][cornerY  ]
                + inputIntPlane[cornerX  ][cornerY  ]);
            
            delta += trCorner * xMinCurrFrac;
            delta += brCorner * xMaxCurrFrac;
            } // if (exact)

            delta += inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMaxCurr +1, w))];
            delta -= inputIntPlane
                [max(0,min(x+xMaxCurr   , h))][max(0,min(y+yMaxCurr   , w))];
            delta -= inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMaxCurr +1, w))];
            delta += inputIntPlane
                [max(0,min(x+xMinCurr   , h))][max(0,min(y+yMaxCurr   , w))];

            delta *= (y+yMaxCurr  >= 0) & (y+yMaxCurr  < w);

            *tmpArray = delta;
        }
    }
}

template <bool exact>
void boxConvAccGradParameters(
    // tmpArray size: {batchSize, nInputPlanes, numFilters, h, w}
    at::Tensor & xMinInt , at::Tensor & xMaxInt , at::Tensor & yMinInt , at::Tensor & yMaxInt ,
    at::Tensor & xMinFrac, at::Tensor & xMaxFrac, at::Tensor & yMinFrac, at::Tensor & yMaxFrac,
    at::Tensor & input_integrated, at::Tensor & tmpArray, Parameter parameter) {

    // TODO switch to square blocks?
    const int threadsNeeded = tmpArray.numel();
    int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(tmpArray.scalar_type(), "gpu::boxConvAccGradParameters", ([&] {
        auto inputIntFlattened = input_integrated.view(
            {-1, input_integrated.size(-2), input_integrated.size(-1)});
        auto inputIntAcsr =
            inputIntFlattened.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        switch (parameter) {
        case Parameter::xMin:
            boxConvAccGradParametersKernel <Parameter::xMin, exact>
                <<<numBlocks, NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, tmpArray.data_ptr<scalar_t>(),
                xMinInt.data_ptr<int32_t>(),  xMaxInt.data_ptr<int32_t>(),
                yMinInt.data_ptr<int32_t>(),  yMaxInt.data_ptr<int32_t>(),
                xMinFrac.data_ptr<scalar_t>(), xMaxFrac.data_ptr<scalar_t>(),
                yMinFrac.data_ptr<scalar_t>(), yMaxFrac.data_ptr<scalar_t>(), xMinInt.numel()); break;
        case Parameter::xMax:
            boxConvAccGradParametersKernel <Parameter::xMax, exact>
                <<<numBlocks, NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, tmpArray.data_ptr<scalar_t>(),
                xMinInt.data_ptr<int32_t>(),  xMaxInt.data_ptr<int32_t>(),
                yMinInt.data_ptr<int32_t>(),  yMaxInt.data_ptr<int32_t>(),
                xMinFrac.data_ptr<scalar_t>(), xMaxFrac.data_ptr<scalar_t>(),
                yMinFrac.data_ptr<scalar_t>(), yMaxFrac.data_ptr<scalar_t>(), xMinInt.numel()); break;
        case Parameter::yMin:
            boxConvAccGradParametersKernel <Parameter::yMin, exact>
                <<<numBlocks, NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, tmpArray.data_ptr<scalar_t>(),
                xMinInt.data_ptr<int32_t>(),  xMaxInt.data_ptr<int32_t>(),
                yMinInt.data_ptr<int32_t>(),  yMaxInt.data_ptr<int32_t>(),
                xMinFrac.data_ptr<scalar_t>(), xMaxFrac.data_ptr<scalar_t>(),
                yMinFrac.data_ptr<scalar_t>(), yMaxFrac.data_ptr<scalar_t>(), xMinInt.numel()); break;
        case Parameter::yMax:
            boxConvAccGradParametersKernel <Parameter::yMax, exact>
                <<<numBlocks, NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, tmpArray.data_ptr<scalar_t>(),
                xMinInt.data_ptr<int32_t>(),  xMaxInt.data_ptr<int32_t>(),
                yMinInt.data_ptr<int32_t>(),  yMaxInt.data_ptr<int32_t>(),
                xMinFrac.data_ptr<scalar_t>(), xMaxFrac.data_ptr<scalar_t>(),
                yMinFrac.data_ptr<scalar_t>(), yMaxFrac.data_ptr<scalar_t>(), xMinInt.numel()); break;
        }
        THCudaCheck(hipGetLastError());
    }));
}

// explicitly instantiate
template void boxConvAccGradParameters<true>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, Parameter);

template void boxConvAccGradParameters<false>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, Parameter);

}
